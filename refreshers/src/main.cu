#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipError_t error;
    int deviceCount;
    
    error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
        return -1;
    }
    
    printf("Number of CUDA devices: %d\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        error = hipGetDeviceProperties(&deviceProp, i);
        
        if (error != hipSuccess) {
            printf("Error: %s\n", hipGetErrorString(error));
            return -1;
        }
        
        printf("\nDevice %d: %s\n", i, deviceProp.name);
        printf("  Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Max Grid Size: (%d, %d, %d)\n", 
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("  Max Block Size: (%d, %d, %d)\n", 
               deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("  Max Threads per Block: %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Max Threads per Multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("  Number of Multiprocessors: %d\n", deviceProp.multiProcessorCount);
        printf("  Warp Size: %d\n", deviceProp.warpSize);
    }
    
    return 0;
}
